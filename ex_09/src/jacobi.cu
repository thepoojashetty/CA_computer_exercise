#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>
#include <time.h>
#include <sys/time.h>

const int n = 134217728; // no of double elements in the array


uint64_t get_time_us(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (uint64_t) (((double) a.tv_nsec / 1000.0) + ((double) a.tv_sec * (1000.0 * 1000.0)));
}

__global__ void update_grid(double *src, const double *tgt, int width, int height) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    if ((tidx < width) && (tidy < height)) {
        ;
    }
}
__device__ unsigned long long get_time_us_device() {
    return clock64();
}

// Host function to call the device function
unsigned long long get_time_us_host() {
    unsigned long long result;
    hipMemcpyFromSymbol(&result, HIP_SYMBOL(get_time_us_device), sizeof(unsigned long long));
    return result;
}

int main() {
    uint64_t start, end,size;
    // Allocate and initialize arrays B and C on the CPU
    printf("size of n");
    printf("%d\n", n);

    size=n*sizeof(double);
    // Allocate memory on the host
    double *gridSrcHst = (double*) _mm_malloc(size,64);
    double *gridTgtHst = (double*) _mm_malloc(size,64);
    int X,Y;
    X = int(sqrt(n));
    Y=X;
    for(uint32_t i=0;i<Y;i++){
        for(uint32_t j=0;j<X;j++){
            if (i == 0 || j == 0){
                gridSrcHst[i * X + j] = gridTgtHst[i * X + j] = 1.0;
            }
            else{
                gridSrcHst[i * X + j] = gridTgtHst[i * X + j] = 0.0;
            }
        }
    }

    double *gridSrcDev, *gridTgtDev; 
    hipMalloc((void**)&gridSrcDev, size);
    hipMalloc((void**)&gridTgtDev, size);

    start = get_time_us();
    hipMemcpy(gridSrcDev, gridSrcHst, size, hipMemcpyHostToDevice);
    hipMemcpy(gridTgtDev, gridTgtHst, size, hipMemcpyHostToDevice);

    end = get_time_us();
    double toGPU_s = ((double)(end - start))/1000000.0;

    printf("Copying data to gpu took: %lf seconds\n", toGPU_s);

    int threadsPerBlock = 512;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    // Launch the STREAM Triad kernel
    start = get_time_us();
    //streamTriad<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, c, n);
    hipDeviceSynchronize();
    end = get_time_us();
    double runKernel_s = ((double)(end - start))/1000000.0;
    printf("Running kernel took: %lf seconds\n", runKernel_s);

    // Copy the results back to the host
    start = get_time_us();
    hipMemcpy(gridTgtHst, gridTgtDev, size, hipMemcpyDeviceToHost);
    end = get_time_us();
    double toCPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to cpu took: %lf seconds\n", toCPU_s);

    // Print the results
    printf("gridTgtHst[0] = %f\n", gridTgtHst[0]);
    printf("gridTgtHst[n-1] = %f\n", gridTgtHst[n-1]);

    // Compute the bandwidth
    printf("Internal memory GPU bandwidth : %lf GB/s\n", (double)3/runKernel_s);
    printf("External memory GPU bandwidth : %lf GB/s\n", (double)3/(toGPU_s+runKernel_s+toCPU_s));

    // Free allocated memory on the GPU
    hipFree(gridTgtDev);
    hipFree(gridSrcDev);

    // Free allocated memory on the CPU
    free(gridTgtHst);
    free(gridSrcHst);
    return 0;
}
