#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>
#include <time.h>
#include <sys/time.h>

uint64_t get_time_us(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (uint64_t) (((double) a.tv_nsec / 1000.0) + ((double) a.tv_sec * (1000.0 * 1000.0)));
}

// CUDA kernel for jacobi iteration on the GPU
__global__ void jacobi(double* grid_source, double* grid_target, uint32_t *_X, uint32_t *_Y) {
    uint32_t X = *_X;
    uint32_t Y = *_Y;
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x>0 && x<X-1 && y>0 && y<Y-1) {
        grid_target[y*X+x] = 0.25 * (grid_source[(y-1)*X+x] + grid_source[(y+1)*X+x] + grid_source[y*X+(x-1)] + grid_source[y*X+(x+1)]);
    }
}

int main() {
    uint64_t start, end;
    uint32_t sizeGib = 3;
    uint64_t sizeBytes = sizeGib*1024*1024*1024;
    uint32_t X = sqrt(sizeBytes/sizeof(double));
    uint32_t Y = X;
    uint32_t size = X*Y;
    uint64_t minimum_runtime_ms = 100;
    uint64_t minimal_runtime_us = minimum_runtime_ms * 1000u;

    printf("Minimum runtime: %" PRIu64 " ms\n", minimum_runtime_ms);

    // Allocate memory on the host
    double *grid_source = (double*) _mm_malloc(size*sizeof(double),64);
    double *grid_target = (double*) _mm_malloc(size*sizeof(double),64);

    double *d_grid_source, *d_grid_target;
    hipMalloc((void**)&d_grid_source, size*sizeof(double));
    hipMalloc((void**)&d_grid_target, size*sizeof(double));

    uint32_t *d_X, *d_Y;
    hipMalloc((void**)&d_X, sizeof(uint32_t));
    hipMalloc((void**)&d_Y, sizeof(uint32_t));

    int threadsPerBlockX = 32;
    int threadsPerBlockY = 32;

    dim3 threadsPerBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 blocksPerGrid((X + threadsPerBlockX - 1) / threadsPerBlockX, (Y + threadsPerBlockY - 1) / threadsPerBlockY);


    start = get_time_us();
    // cudaMemcpy(d_A, A, GB, cudaMemcpyHostToDevice);
    hipMemcpy(d_grid_source, grid_source, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_target, grid_target, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_X, &X, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, &Y, sizeof(uint32_t), hipMemcpyHostToDevice);
    end = get_time_us();
    double toGPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to gpu took: %lf seconds\n", toGPU_s);


    // Launch the jacobi kernel
    uint64_t actual_runtime = 0u;
    uint64_t runs           = 0u;
    double *temp;
    for(runs = 1u; actual_runtime < minimal_runtime_us; runs = runs << 1u) {
		for(uint32_t i=0;i<Y;i++){
			for(uint32_t j=0;j<X;j++){
				if (i == 0 || j == 0){
					grid_source[i * X + j] = grid_target[i * X + j] = 1.0;
				}
				else{
					grid_source[i * X + j] = grid_target[i * X + j] = 0.0;
				}
			}
		}
		start = get_time_us();
		for(uint64_t i = 0u; i < runs; i++) {
			jacobi<<<blocksPerGrid, threadsPerBlock>>>(d_grid_source, d_grid_target, d_X, d_Y);
            hipDeviceSynchronize();
			//swap source and target
			temp=d_grid_source;
			d_grid_source=d_grid_target;
			d_grid_target=temp;
		}
		end = get_time_us();
		actual_runtime = end - start;
	}
    runs = runs>>1u;
    uint32_t n=(X-2)*(Y-2);
    double runKernel_s = ((double)actual_runtime)/1000000.0;
    printf("Running kernel took: %lf seconds\n", runKernel_s);

    // Copy the results back to the host
    start = get_time_us();
    hipMemcpy(grid_target, d_grid_source, size*sizeof(double), hipMemcpyDeviceToHost);
    end = get_time_us();
    double toCPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to cpu took: %lf seconds\n", toCPU_s);

    // Compute the bandwidth
    printf("Internal memory GPU bandwidth : %lf GB/s\n", (double)(runs*6)/runKernel_s);
    printf("External memory GPU bandwidth : %lf GB/s\n", (double)(runs*6)/(toGPU_s+runKernel_s+toCPU_s));

    // Free allocated memory on the GPU
    hipFree(d_grid_source);
    hipFree(d_grid_target);

    // Free allocated memory on the CPU
    free(grid_source);
    free(grid_target);
    return 0;
}
