#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>
#include <time.h>
#include <sys/time.h>

uint64_t get_time_us(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (uint64_t) (((double) a.tv_nsec / 1000.0) + ((double) a.tv_sec * (1000.0 * 1000.0)));
}
// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}

// CUDA kernel for jacobi iteration on the GPU
__global__ void matrixMul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}

int main() {

    uint64_t start, end;
    uint32_t sizeGiB = 2.6;
    uint64_t sizeBytes = sizeGiB*1024*1024*1024;
    uint32_t X = sqrt(sizeBytes/sizeof(double));
    uint32_t Y = X;
    uint32_t size = X*Y;
    uint64_t minimum_runtime_ms = 100;

    double *h_A = (double *)_mm_malloc(size*sizeof(double),64);
    double *h_B = (double *)_mm_malloc(size*sizeof(double),64);
    double *h_C = (double *)_mm_malloc(size*sizeof(double),64);
    printf("Minimum runtime: %" PRIu64 " ms\n", minimum_runtime_ms);

    double *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size*sizeof(double));
    hipMalloc((void**)&d_B, size*sizeof(double));
    hipMalloc((void**)&d_C, size*sizeof(double));
    GPU_fill_rand(d_A, X, Y);
    GPU_fill_rand(d_B, X, Y);

    start = get_time_us();
    // hipMemcpy(d_A, A, GB, hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size*sizeof(double), hipMemcpyHostToDevice);
    end = get_time_us();
    double toGPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to gpu took: %lf seconds\n", toGPU_s);

    int threadsPerBlockX = 32;
    int threadsPerBlockY = 32;

    dim3 threadsPerBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 blocksPerGrid((X + threadsPerBlockX - 1) / threadsPerBlockX, (Y + threadsPerBlockY - 1) / threadsPerBlockY);
    uint64_t actual_runtime = 0u;
    start = get_time_us();
    // Launch the matrix multiplication kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, X, Y, Y);
    hipDeviceSynchronize();
    end = get_time_us();
    actual_runtime = end - start;

    /*uint64_t actual_runtime = 0u;
    uint64_t runs           = 0u;
    for(runs = 1u; actual_runtime < minimal_runtime_us; runs = runs << 1u) {
		start = get_time_us();
		for(uint64_t i = 0u; i < runs; i++) {
			matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, X, Y, Y);
            hipDeviceSynchronize();
		}
		end = get_time_us();
		actual_runtime = end - start;
	}
    runs = runs>>1u;
    uint32_t n=(X-2)*(Y-2);*/
    double runKernel_s = ((double)actual_runtime)/1000000.0;
    printf("Running kernel took: %lf seconds\n", runKernel_s);

    // Copy the results back to the host
    start = get_time_us();
    hipMemcpy(h_C, d_C, size*sizeof(double), hipMemcpyDeviceToHost);
    end = get_time_us();
    double toCPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to cpu took: %lf seconds\n", toCPU_s);

    // Compute the bandwidth
    printf("Internal memory GPU bandwidth : %lf GB/s\n", (double)3/runKernel_s);
    printf("External memory GPU bandwidth : %lf GB/s\n", (double)3/(toGPU_s+runKernel_s+toCPU_s));

    // Free allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free allocated memory on the CPU
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
