#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>
#include <time.h>
#include <sys/time.h>

uint64_t get_time_us(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (uint64_t) (((double) a.tv_nsec / 1000.0) + ((double) a.tv_sec * (1000.0 * 1000.0)));
}
// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}

// CUDA kernel for cublas matrix multiplication
void matrixMul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}

int main() {
    uint64_t sizeBytes = 2.6*1024*1024*1024;
    uint32_t X = sqrt(sizeBytes/sizeof(double));
    uint32_t Y = X;
    uint32_t size = X*Y;
    uint64_t runtime = 0u;
    uint64_t start, end;

    double *h_A = (double *)_mm_malloc(size*sizeof(double),64);
    double *h_B = (double *)_mm_malloc(size*sizeof(double),64);
    double *h_C = (double *)_mm_malloc(size*sizeof(double),64);

    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size*sizeof(double));
    hipMalloc((void**)&d_B, size*sizeof(double));
    hipMalloc((void**)&d_C, size*sizeof(double));

    GPU_fill_rand(d_A, X, Y);
    GPU_fill_rand(d_B, X, Y);

    start = get_time_us();
    // Launch the matrix multiplication
    matrixMul(d_A, d_B, d_C, X, Y, Y);
    end = get_time_us();
    runtime = end - start;

    double runtime_s = ((double)runtime)/1000000.0;
    printf("Running matmul took: %lf seconds\n", runtime_s);

    // Copy the results back to the host
    // start = get_time_us();
    hipMemcpy(h_C, d_C, size*sizeof(double), hipMemcpyDeviceToHost);
    // end = get_time_us();
    // double toCPU_s = ((double)(end - start))/1000000.0;
    // printf("Copying data to cpu took: %lf seconds\n", toCPU_s);

    // Compute the bandwidth
    printf("Floating point performance with GPU : %lf Flops/s\n", (double)(2*pow(X,3))/runtime_s);

    // Free allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free allocated memory on the CPU
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
