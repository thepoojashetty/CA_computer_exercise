#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>


const int n = 2147483648; // 2 GiB per array
const double c = 2.0f; // Arbitrary constant (unequal to {0.0, 1.0})

// CUDA kernel for STREAM Triad
__global__ void streamTriad(double *A, const double *B, const double *C, const double c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        A[tid] = B[tid] * c + C[tid];
    }
}
__device__ unsigned long long get_time_us_device() {
    return clock64();
}

// Host function to call the device function
unsigned long long get_time_us_host() {
    unsigned long long result;
    hipMemcpyFromSymbol(&result, HIP_SYMBOL(get_time_us_device), sizeof(unsigned long long));
    return result;
}

int main() {
    // Allocate and initialize arrays B and C on the CPU
    /*unsigned long long start = 0u;
	unsigned long long stop  = 0u;
    unsigned long long runtime = 0u;*/
    /*float *h_B = new float[n];
    float *h_C = new float[n];*/
    printf("size of n");
    printf("%" PRId64 "\n", n);

    // Allocate memory on the host
    double *h_B = (double*) _mm_malloc(n*sizeof(double),64);
    double *h_C = (double*) _mm_malloc(n*sizeof(double),64);
    for (int i = 0; i < n; ++i) {
        h_B[i] = static_cast<double>(i);
        h_C[i] = static_cast<double>(n - i);
    }
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, n*sizeof(double));
    hipMalloc((void**)&d_B, n*sizeof(double));
    hipMalloc((void**)&d_C, n*sizeof(double));
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float dataTxTime = 0;
    float execTime = 0;
    float totalExecTime = 0;
    // Copy arrays B and C from host to device
    hipEventRecord(start);
    hipMemcpy(d_B, h_B, n*sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, n*sizeof(double) , hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    dataTxTime+=milliseconds;

    printf("Host to device data transfer time: %f ms\n",milliseconds);

    // Configure kernel launch parameters
    /*int blockSize = 512; //512 threads per block
    int gridSize = (n + blockSize - 1) / blockSize;*/
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n*sizeof(double) + threadsPerBlock.x -1) / threadsPerBlock.x, (n*sizeof(double)+threadsPerBlock.y -1) / threadsPerBlock.y);
    // Launch the STREAM Triad kernel

    // Record start time
    hipEventRecord(start);
    printf("Running kernel now!\n");
    streamTriad<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, c, n);
    printf("done executing kernel !\n");
    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    execTime+=milliseconds;
    totalExecTime = execTime+dataTxTime;
    printf("Kernel execution time: %f ms\n",execTime);
    printf("Total execution time: %f ms\n",totalExecTime);
    // Copy array A from device to host for verification
    hipMemcpy(h_B, d_A, n, hipMemcpyDeviceToHost);

    // Free allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Free allocated memory on the CPU
    delete[] h_B;
    delete[] h_C;

    return 0;
}
