#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <inttypes.h>
#include <sys/stat.h>
#include <malloc.h>
#include <unistd.h>
#include <immintrin.h>
#include <time.h>
#include <sys/time.h>

const int n = 134217728; // no of double elements in the array
const double c = 2.0f; // Arbitrary constant (unequal to {0.0, 1.0})

uint64_t get_time_us(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (uint64_t) (((double) a.tv_nsec / 1000.0) + ((double) a.tv_sec * (1000.0 * 1000.0)));
}

// CUDA kernel for STREAM Triad
__global__ void streamTriad(double *A, const double *B, const double *C, const double c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        A[tid] = B[tid] * c + C[tid];
    }
}
__device__ unsigned long long get_time_us_device() {
    return clock64();
}

// Host function to call the device function
unsigned long long get_time_us_host() {
    unsigned long long result;
    hipMemcpyFromSymbol(&result, HIP_SYMBOL(get_time_us_device), sizeof(unsigned long long));
    return result;
}

int main() {
    uint64_t start, end,size;
    // Allocate and initialize arrays B and C on the CPU
    printf("size of n");
    printf("%d\n", n);

    size=n*sizeof(double);
    // Allocate memory on the host
    double *h_A = (double*) _mm_malloc(size,64);
    double *h_B = (double*) _mm_malloc(size,64);
    double *h_C = (double*) _mm_malloc(size,64);
    for (int i = 0; i < n; ++i) {
        h_B[i] = 1;
        h_C[i] = 1;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    start = get_time_us();
    // hipMemcpy(d_A, A, GB, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    end = get_time_us();
    double toGPU_s = ((double)(end - start))/1000000.0;

    printf("Copying data to gpu took: %lf seconds\n", toGPU_s);

    int threadsPerBlock = 512;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    // Launch the STREAM Triad kernel
    start = get_time_us();
    streamTriad<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, c, n);
    hipDeviceSynchronize();
    end = get_time_us();
    double runKernel_s = ((double)(end - start))/1000000.0;
    printf("Running kernel took: %lf seconds\n", runKernel_s);

    // Copy the results back to the host
    start = get_time_us();
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    end = get_time_us();
    double toCPU_s = ((double)(end - start))/1000000.0;
    printf("Copying data to cpu took: %lf seconds\n", toCPU_s);

    // Print the results
    printf("A[0] = %f\n", h_A[0]);
    printf("A[n-1] = %f\n", h_A[n-1]);

    // Compute the bandwidth
    printf("Internal memory GPU bandwidth : %lf GB/s\n", (double)3/runKernel_s);
    printf("External memory GPU bandwidth : %lf GB/s\n", (double)3/(toGPU_s+runKernel_s+toCPU_s));

    // Free allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free allocated memory on the CPU
    free(h_B);
    free(h_C);
    free(h_A);
    return 0;
}
